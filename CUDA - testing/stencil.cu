#include <hip/hip_runtime.h>

#define N (2048 * 2048)
#define BLOCK_SIZE 264
#define RADIUS 3

__global__ void stencil_1d(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;

	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS)
	{
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}

	__syncthreads();

	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++)
	{
		result += temp[lindex + offset];
	}

	out[gindex] = result;
}

int main() {}